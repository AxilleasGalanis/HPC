#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"

void checkCudaErrors(hipError_t error){
	if(error != hipSuccess) {
		printf("\033[0;31mCUDA Error: %s in %s, line %d\033[0;37m\n", hipGetErrorString(error), __FILE__, __LINE__);
	}
}

PGM_IMG contrast_enhancement_g(PGM_IMG img_in)
{
    PGM_IMG result;
    int *hist, *lut;
    hipEvent_t start, stop;
    unsigned char *source_img, *result_img;
    result.w = img_in.w;
    result.h = img_in.h;

	result.img = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));
	hipMalloc((void **)&result_img, img_in.h * img_in.w * sizeof(unsigned char));
    hipMalloc((void **)&source_img, img_in.h * img_in.w * sizeof(unsigned char));
    hipMalloc((void **)&hist, 256 * sizeof(int));
    hipMalloc((void **)&lut, 256 * sizeof(int));
    hipEventCreate(&start);
    hipEventCreate(&stop);


    dim3 block_dim(16, 16);
    dim3 grid_dim(ceil((double)img_in.w / 16), ceil((double)img_in.h / 16));
    
	//dim3 threads( 16, 4, 1 );
	//int numthreads = threads.x*threads.y;
    //int numblocks = ceil((double)(img_in.w*img_in.h) / (numthreads*255)) ;

    hipMemset(hist, 0, 256*sizeof(int));
    checkCudaErrors(hipGetLastError());
    hipMemcpy(source_img, img_in.img, img_in.h * img_in.w * sizeof(unsigned char), hipMemcpyHostToDevice);
    checkCudaErrors(hipGetLastError());

	hipEventRecord(start);
    d_histogram<<<600, 256>>>(hist, source_img, img_in.h * img_in.w);
    checkCudaErrors(hipGetLastError());
    d_lut<<<1, 256>>>(lut, hist, img_in.h * img_in.w);
	checkCudaErrors(hipGetLastError());

    d_histogram_result<<<grid_dim, block_dim>>>(result_img, source_img, lut, img_in.w*img_in.h);
	checkCudaErrors(hipGetLastError());
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float diff = 0;
    hipEventElapsedTime(&diff, start, stop);
    
    hipMemcpy(result.img, result_img, img_in.h * img_in.w * sizeof(unsigned char), hipMemcpyDeviceToHost);
	checkCudaErrors(hipGetLastError());
	hipDeviceSynchronize();

    hipFree(result_img);
    hipFree(source_img);
    hipFree(hist);
    hipFree(lut);
    hipEventDestroy(start);
	hipEventDestroy(stop);

    hipDeviceReset();
	printf("GPU time = %.10f seconds\n", diff / 1000);
    return result;
}
